#include "hip/hip_runtime.h"
/**
 * PaSS.cu
 * The main functions of PaSS
 *
 * @author emfo
 */

#include "PaSS_BLAS.cu"
using namespace pass_blas;

#include <algorithm>
using namespace std;

/**
 * The PaSS namespace
 */
namespace pass{
	/**
	 * The status enumeration
	*/
	enum Stat {
		init = 0,  /**< initialing */
		forw = 1,  /**< going forward */
		back = -1, /**< going backward */
	};

	/**
	 * The criterion enumeration
	 */
	enum Criterion {
		AIC,   /**< Akaike information criterion */
		BIC,   /**< Bayesian information criterion */
		EBIC,  /**< EBIC */
		HDBIC, /**< HDBIC */
		HDHQ   /**< HDHQ */
	};

	/**
	 * The parameter structure
	 */
	struct Parameter {
		u32 nP;    /**< the number of particle */
		u32 nI;    /**< the number of iteration */
		float pfg; /**< the probability for forward step: global */
		float pfl; /**< the probability for forward step: local */
		float pfr; /**< the probability for forward step: random */
		float pbl; /**< the probability for backward step: local */
		float pbr; /**< the probability for backward step: random */
	};

	/**
	 * The data structure
	 */
	struct Data {
		vec* Beta;  /**< the vector beta */
		float e;    /**< the norm of R */
		idx* Index; /**< the index of chosen column of X */
		mat* InvA;  /**< the inverse of A */
		float phi;  /**< the value given by criterion */
		vec* R;     /**< the difference between Y and Beta */
		Stat stat;  /**< the status */
		vec* Theta; /**< the vector theta */
		mat* X;     /**< the data we chosen */
	};
}
using namespace pass;


// Global variables
__device__ u32 n, p;
__device__ mat* X;
__device__ vec* Y;
__device__ Criterion cri = HDBIC;
__device__ Parameter par = {16, 128, .8, .1, .1, .9, .1};
__device__ Data* data_best;
__device__ hiprandState s;

// Functions
void pass_init(float*, float*, const u32, const u32);
hipError_t pass_host(const float*, const float*, u32*, const u32, const u32);
__global__ void pass_kernel(const float*, const float*, u32*, const u32, const u32);
__device__ bool pass_update_fb(Data*);
__device__ bool pass_update_cri(Data*, const u32);


/**
 * PaSS main function
 */
int main() {
	// Declare variables
	u32 host_n = 100;
	u32 host_p = 20;
	float* host_X = (float*)malloc(host_n * host_p * sizeof(float));
	float* host_Y = (float*)malloc(host_n * sizeof(float));
	u32 *host_I = (u32*)malloc(host_p * sizeof(u32));
	
	// Initialize data
	pass_init(host_X, host_Y, host_n, host_p);

	// Display data
	//u32 i, j;
	//printf("X:\n");
	//for(i = 0; i < host_n; i++) {
	//	for(j = 0; j < host_p; j++) {
	//		printf("%8.3f  ", host_X[i*host_p + j]);
	//	}
	//	printf("\n");
	//}
	//printf("\n\nY:\n");
	//for(i = 0; i < host_n; i++) {
	//	printf("%8.3f  ", host_Y[i]);
	//}
	//printf("\n\n");
	
	// Run PaSS
	pass_host(host_X, host_Y, host_I, host_n, host_p);
	
	system("pause");
	return 0;
}


/**
 * Initialize data
 *
 * @param host_X the matrix X
 * @param host_Y the vector Y
 * @param host_n the number of rows in X
 * @param host_p the number of columns in X
 */
void pass_init(float* host_X, float* host_Y, const u32 host_n, const u32 host_p) {
	u32 i, j;
	for(i = 0; i < host_n; i++) {
		for(j = 0; j < host_p; j++) {
			host_X[i*host_p + j] = (float)(i+1)*(2*j+1);
		}
		host_Y[i] = (float)10*i;
	}
}


/**
 * PaSS host function
 *
 * @param host_X the matrix X
 * @param host_Y the vector Y
 * @param host_n the number of rows in X
 * @param host_p the number of columns in X
 */
hipError_t pass_host(const float* host_X, const float* host_Y, u32* host_I, const u32 host_n, const u32 host_p) {
	// Declare variables
	float* dev_X = 0;
	float* dev_Y = 0;
	u32 *dev_I = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!	Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	
	// Allocate GPU buffers for data (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_X, host_n * host_p * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Y, host_n * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_I, host_n * sizeof(u32));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_X, host_X, host_n * host_p * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Y, host_Y, host_n * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch the kernel function on the GPU with one thread for each element.
	pass_kernel<<<1, 1>>>(dev_X, dev_Y, dev_I, host_n, host_p);

	// Check for any errors launching the kernel.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(host_I, dev_I, host_p * sizeof(u32), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_X);
	hipFree(dev_Y);
	hipFree(dev_I);
	return cudaStatus;
}


/**
 * PaSS kernel function
 *
 * @param array_X the matrix X
 * @param array_Y the vector Y
 * @param host_n the number of rows in X
 * @param host_p the number of columns in X
 */
__global__ void pass_kernel(const float* array_X, const float* array_Y, u32* array_I, const u32 host_n, const u32 host_p) {
	// Initialize Random Seed
	hiprand_init(clock64(), 0, 0, &s);

	// Declare variables
	n = host_n;
	p = host_p;
	X = new mat(n, p);
	Y = new vec(n);
	u32 i, j;
	
	// Copy X and Y from array to matrix
	for(j = 0; j < p; j++) {
		for(i = 0; i < n; i++) {
			X->col[j]->e[i] = array_X[i*p + j];
		}
	}
	for(u32 i = 0; i < n; i++) {
		Y->e[i] = array_Y[i];
	}

	// Set Random Initial
	bool isRandInitial = false;


	// Initialize Particles
	Data* data = new Data[par.nP];
	mat* Phi = new mat(par.nI+1, par.nP);
	if(isRandInitial || p < par.nP) {
		for(j = 0; j < par.nP; j++) {
			data[j].stat = init;
			pass_update_cri(&data[j], hiprand(&s) % p);
			Phi->col[j]->e[0] = data[j].phi;
			data[j].stat = forw;
		}
	}
	else {
		vec* C = new vec(p);
		idx* I = new idx(p);
		for(j = 0; j < par.nP; j++) {
			mul(C, Y, X);
			sort_index_descend(I, C);
			data[j].stat = init;
			pass_update_cri(&data[j], I->e[j]);
			Phi->col[j]->e[0] = data[j].phi;
			data[j].stat = forw;
		}
		delete C;
		delete I;
	}
	
	// Choose Global Best
	data_best = &data[0];

	// Find Best Data
	for(i = 0; i < par.nI; i++) {
		for(j = 0; j < par.nP; j++) {
			pass_update_fb(&data[j]);
			Phi->col[j]->e[i+1] = data[j].phi;
			if(data_best->phi > data[j].phi) {
				data_best = &data[j];
			}
			if(data[j].phi - Phi->col[j]->e[i] > 0) {
				data[j].stat = (Stat)(-data[j].stat);
			}
			if(data[j].Index->n <= 1) {
				data[j].stat = forw;
			}
			if(data[j].Index->n >= p - 5) {
				data[j].stat = back;
			}
		}
	}
	
	delete X;
	delete Y;
}


/**
 * Determine forward or backward
 */
__device__ bool pass_update_fb(Data* data) {
	u32 index = 0;

	switch(data->stat) {
	case forw: // Forward
		{
			idx* Index_B = new idx(data_best->Index->n);
			idx* Index_C = new idx(data->Index->n);
			idx* Index_D = new idx(Index_B->n);
			idx* Index_R = new idx(p - Index_C->n);

			// Sort Index_B
			copy(Index_B, data_best->Index);
			sort_ascend(Index_B);
			
			// Sort Index_C
			copy(Index_C, data->Index);
			sort_ascend(Index_C);
			
			// Let Index_D be Index_B exclude Index_C
			set_difference(Index_D, Index_B, Index_C);
			
			// Let Index_R be the complement of Index_C
			complement(Index_R, Index_C, p);

			// Determine the index to add
			if(hiprand_uniform(&s) < par.pfg && Index_D->n > 0){
				index = Index_D->e[hiprand(&s) % Index_D->n];
			}
			else if(hiprand_uniform(&s) < par.pfl/(par.pfl+par.pfr)){
				float phi_max = -1, phi_temp;
				for(u32 i = 0; i < Index_R->n; i++) {
					inner(&phi_temp, data->R, X->col[i]);
					phi_temp = abs(phi_temp);
					if(phi_temp > phi_max) {
						phi_max = phi_temp;
						index = Index_R->e[i];
					}
				}
			}
			else{
				index = Index_R->e[hiprand(&s) % Index_R->n];
			}
			
			delete Index_B;
			delete Index_C;
			delete Index_D;
			delete Index_R;
			break;
		}
	case back: // Backward
		{
			// Determine the index to remove
			if(hiprand_uniform(&s)< par.pbl){
				mat* B = new mat(data->X->n_row, data->X->n_col);
				vec* C = new vec(data->X->n_col);
				u32 ii;
				for(u32 i = 0; i < B->n_col; i++){
					mul(B->col[i], data->X->col[i], data->Beta->e[i]);
					add(B->col[i], B->col[i], data->R);
				}
				inner(C, B);
				find_min_index(&ii, C);
				index = data->Index->e[ii];
				delete B;
				delete C;
			}
			else{
				index = data->Index->e[hiprand(&s) % data->Index->n];
			}
			break;
		}
	}
	pass_update_cri(data, index);
	return true;
}


/**
 * Compute the value given by criterion
 *
 * @param data the updating data
 * @param index the index to compute
 */
__device__ bool pass_update_cri(Data* data, const u32 index) {
	float gamma = 1;
	u32 k = 0;
	vec* Xnew = X->col[index];
	switch(data->stat) {
	case init: // Initial
		{
			data->X = new mat(n, 1);
			copy(data->X->col[0], Xnew);

			data->InvA = new mat(1, 1);
			float a;
			inner(&a, Xnew);
			data->InvA->col[0]->e[0] = 1 / a;

			data->Theta = new vec(1);
			inner(data->Theta->e, Xnew, Y);

			data->Beta = new vec(1);
			mul(data->Beta, data->InvA, data->Theta);

			data->Index = new idx(1);
			data->Index->e[0] = index;

			data->R = new vec(n);

			k = 1;
		}
		break;
	case forw: // Forward
		{
			k = data->Index->n;
			vec* B = new vec(k);
			vec* D = new vec(k);
			mat* InvAtemp = new mat(k+1, k+1);
			float alpha;
			float c1;
			float c2;

			mul(B, Xnew, data->X);

			mul(D, data->InvA, B);
			
			inner(&c1, Xnew);
			inner(&c2, B, D);
			alpha = 1/(c1 - c2);

			insert(D, -1);

			insert_col(data->X, Xnew);

			mul(InvAtemp, D, D);
			mul(InvAtemp, InvAtemp, alpha);
			insert(data->InvA, 0);
			add(data->InvA, data->InvA, InvAtemp);
			
			inner(&c1, Xnew, Y);
			insert(data->Theta, c1);
			
			inner(&c2, D, data->Theta);
			mul(D, D, alpha*c2);
			insert(data->Beta, 1);
			add(data->Beta, data->Beta, D);

			insert(data->Index, index);
			
			delete B;
			delete D;
			delete InvAtemp;
			k++;
		}
		break;
	case back: // Backward
		{
			k = data->Index->n - 1;
			u32 ii;
			mat* E = new mat(k, k);
			vec* F = new vec(k);
			float g;

			find_index(&ii, data->Index, index);
			if(ii != k) {
				swap_col(data->X, ii, k);
				swap(data->Theta, ii, k);
				swap(data->Beta, ii, k);
				swap_row(data->InvA, ii, k);
				swap_col(data->InvA, ii, k);
				swap(data->Index, ii, k);
			}

			shed_col(data->X);
			shed(data->Theta);
			shed(data->Index);

			g = data->InvA->col[k]->e[k];

			shed_row(data->InvA);
			copy(F, data->InvA->col[k]);
			shed_col(data->InvA);

			mul(E, F, F);
			mul(E, E, 1/g);
			sub(data->InvA, data->InvA, E);

			
			mul(F, F, data->Beta->e[k] / g);
			shed(data->Beta);
			add(data->Beta, data->Beta, F);

			delete E;
			delete F;
		}
		break;
	}

	mul(data->R, data->X,  data->Beta);
	sub(data->R, Y, data->R);

	norm(&data->e, data->R);

	switch(cri) {
	case AIC:
		data->phi = n * log(data->e * data->e / n) + 2 * k;
		break;
	case BIC:
		data->phi = n * log(data->e * data->e / n) + log((float)n) * k;
		break;
	case EBIC:
		data->phi = n * log(data->e * data->e / n) + log((float)n) * k + 2 * gamma * ((p+.5) * log((float)p) - (k+.5) * log((float)k) - (p-k+.5) * log((float)(p-k)) - .5 * log(2 * HIP_PI_F));
		break;
	case HDBIC:
		data->phi = n * log(data->e * data->e / n) + log((float)n) * log((float)p) * k;
		break;
	case HDHQ:
		data->phi = n * log(data->e * data->e / n) + 2.01 * log(log((float)n)) * log((float)p) * k;
		break;
	}
	return true;
}
