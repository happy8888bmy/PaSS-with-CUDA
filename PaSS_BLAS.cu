#include "hip/hip_runtime.h"
/**
 * PaSS_BLAS.cu
 * The basic linear algebra sub-programs for PaSS
 *
 * @author emfo
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "hiprand/hiprand_kernel.h"
#include "hip/device_functions.h"
#include "float.h"
#include "hip/hip_math_constants.h"
#include "math_functions.h"

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

/**
 * The PaSS_BLAS namespace
 */
namespace pass_blas {
	/**
	 * 32-bit unsigned int.
	 */
	typedef uint32_t u32;

	/**
	 * The vector structure
	 */
	struct vec {
		u32 n;    /**< the length */
		float* e; /**< the array of entries */
		
		/**
		 * Construct a vector.
		 *
		 * @param n the length of the vector.
		 */
		__host__ __device__ vec(const u32 n) {
			this->n = n;
			this->e = (float*)malloc(n * sizeof(float));
		}
		
		/**
		 * Construct a vector and fill it with given value.
		 *
		 * @param n the length of the vector.
		 * @param d the value of entries.
		 */
		__host__ __device__ vec(const u32 n, const float d) {
			this->n = n;
			this->e = (float*)malloc(n * sizeof(float));
			for(u32 i = 0; i < n; i++) {
				this->e[i] = d;
			}
		}
		
		/**
		 * Destruct the vector.
		 */
		__host__ __device__ ~vec() {
			free(this->e);
		}
	};


	/**
	 * The matrix structure
	 */
	struct mat {
		u32 n_row; /**< the number of rows */
		u32 n_col; /**< the number of columns */
		vec** col; /**< the array of columns */
		
		/**
		 * Construct a matrix.
		 *
		 * @param p the number of rows of the vector.
		 * @param q the number of columns of the vector.
		 */
		__host__ __device__ mat(const u32 p, const u32 q) {
			this->n_row = p;
			this->n_col = q;
			this->col = (vec**)malloc(q * sizeof(vec*));
			for(u32 i = 0; i < q; i++) {
				this->col[i] = new vec(p);
			}
		}
		
		/**
		 * Construct a matrix and fill it with given value.
		 *
		 * @param p the number of rows of the vector.
		 * @param q the number of columns of the vector.
		 * @param d the value of entries.
		 */
		__host__ __device__ mat(const u32 p, const u32 q, const float d) {
			this->n_row = p;
			this->n_col = q;
			this->col = (vec**)malloc(q * sizeof(vec*));
			for(u32 i = 0; i < q; i++) {
				this->col[i] = new vec(p, d);
			}
		}
		
		/**
		 * Destruct the matrix.
		 */
		__host__ __device__ ~mat() {
			for(u32 i = 0; i < this->n_col; i++) {
				delete this->col[i];
			}
			free(this->col);
		}
	};


	/**
	 * The index structure
	 */
	struct idx {
		u32 n;  /**< the length */
		u32* e; /**< the array of entries */
		
		/**
		 * Construct a index.
		 *
		 * @param n the length of the index.
		 */
		__host__ __device__ idx(const u32 n) {
			this->n = n;
			this->e = (u32*)malloc(n * sizeof(u32));
		}
		
		/**
		 * Construct a index and fill it with given value.
		 *
		 * @param n the length of the index.
		 * @param d the value of entries.
		 */
		__host__ __device__ idx(const u32 n, const u32 d) {
			this->n = n;
			this->e = (u32*)malloc(n * sizeof(u32));
			for(u32 i = 0; i < n; i++) {
				this->e[i] = d;
			}
		}
		
		/**
		 * Destruct the index.
		 */
		__host__ __device__ ~idx() {
			free(this->e);
		}
	};


	/**
	 * Display the vector.
	 *
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool print(const vec* v) {
		for(u32 i = 0; i < v->n; i++) {
			printf("%8.3f  ", v->e[i]);
		}
		printf("\n");
		return true;
	}


	/**
	 * Display the matrix.
	 *
	 * @param a the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool print(const mat* a) {
		for(u32 j = 0; j < a->n_row; j++) {
			for(u32 i = 0; i < a->n_col; i++) {
				printf("%8.3f  ", a->col[i]->e[j]);
			}
			printf("\n");
		}
		printf("\n");
		return true;
	}


	/**
	 * Display the index.
	 *
	 * @param x the index.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool print(const idx* x) {
		for(u32 i = 0; i < x->n; i++) {
			printf("%4u ", x->e[i]);
		}
		printf("\n");
		return true;
	}


	/**
	 * Copy a vector.
	 *
	 * @param u the new vector.
	 * @param v the original vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool copy(vec* u, const vec* v) {
		if(u->n != v->n) {
			printf("(copy: vector) not aligned!\n");
			return false;
		}
		memcpy(u->e, v->e, v->n * sizeof(float));
		return true;
	}


	/**
	 * Copy a matrix.
	 *
	 * @param c the new matrix.
	 * @param a the original matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool copy(mat* c, const mat* a) {
		if(c->n_col != a->n_col || c->n_row != a->n_row) {
			printf("(copy: matrix) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			memcpy(c->col[i], a->col[i], a->n_row * sizeof(float));
		}
		return true;
	}


	/**
	 * Copy a index.
	 *
	 * @param x the new index.
	 * @param z the original index.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool copy(idx* z, const idx* x) {
		if(z->n != x->n) {
			printf("(copy: index) not aligned!\n");
			return false;
		}
		memcpy(z->e, x->e, x->n * sizeof(u32));
		return true;
	}


	/**
	 * u = v+w.
	 *
	 * @param u the sum vector.
	 * @param v the augend vector.
	 * @param w the addend vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool add(vec* u, const vec* v, const vec* w) {
		if(u->n != v->n || u->n != w->n) {
			printf("(add: vector) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < v->n; i++) {
			u->e[i] = v->e[i] + w->e[i];
		}
		return true;
	}


	/**
	 * c = a+b.
	 *
	 * @param b the sum matrix.
	 * @param c the augend matrix.
	 * @param a the addend matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool add(mat* c, const mat* a, const mat* b) {
		if(c->n_col != a->n_col || c->n_col != b->n_col || c->n_row != a->n_row || c->n_row != b->n_row) {
			printf("(add: matrix) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			for(u32 j = 0; j < a->n_row; j++) {
				c->col[i]->e[j] = a->col[i]->e[j] + b->col[i]->e[j];
			}
		}
		return true;
	}


	/**
	 * u = v+w.
	 *
	 * @param u the difference vector.
	 * @param v the minuend vector.
	 * @param w the subtrahend vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool sub(vec* u, const vec* v, const vec* w) {
		if(u->n != v->n || u->n != w->n) {
			printf("(sub: vector) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < v->n; i++) {
			u->e[i] = v->e[i] - w->e[i];
		}
		return true;
	}


	/**
	 * c = a+b.
	 *
	 * @param b the difference matrix.
	 * @param c the minuend matrix.
	 * @param a the subtrahend matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool sub(mat* c, const mat* a, const mat* b) {
		if(c->n_col != a->n_col || c->n_col != b->n_col || c->n_row != a->n_row || c->n_row != b->n_row) {
			printf("(sub: matrix) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			for(u32 j = 0; j < a->n_row; j++) {
				c->col[i]->e[j] = a->col[i]->e[j] - b->col[i]->e[j];
			}
		}
		return true;
	}


	/**
	 * u = d*v.
	 *
	 * @param u the product vector.
	 * @param v the multiplier vector.
	 * @param d the multiplicand number.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool mul(vec* u, const vec* v, const float d) {
		for(u32 i = 0; i < v->n; i++) {
			u->e[i] = v->e[i] * d;
		}
		return true;
	}


	/**
	 * c = d*a.
	 *
	 * @param c the product matrix.
	 * @param a the multiplier vector.
	 * @param d the multiplicand number.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool mul(mat* c, const mat* a, const float d) {
		for(u32 i = 0; i < a->n_col; i++) {
			for(u32 j = 0; j < a->n_row; j++) {
				c->col[i]->e[j] = a->col[i]->e[j] * d;
			}
		}
		return true;
	}


	/**
	 * u = a*v
	 *
	 * @param u the product vector.
	 * @param a the multiplicand matrix.
	 * @param v the multiplier vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool mul(vec* u, const mat* a, const vec* v) {
		if(u->n != a->n_row || v->n != a->n_col) {
			printf("(mul: matrix left) not aligned!\n");
			return false;
		}
		for(u32 j = 0; j < a->n_row; j++) {
			u->e[j] = 0;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			for(u32 j = 0; j < a->n_row; j++) {
				u->e[j] += a->col[i]->e[j] * v->e[i];
			}
		}
		return true;
	}


	/**
	 * u' = v'*a (u = a'*v)
	 *
	 * @param u the product vector.
	 * @param v the multiplicand vector.
	 * @param a the multiplier matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool mul(vec* u, const vec* v, const mat* a) {
		if(u->n != a->n_col || v->n != a->n_row) {
			printf("(mul: matrix right) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			u->e[i] = 0;
			for(u32 j = 0; j < a->n_row; j++) {
				u->e[i] += a->col[i]->e[j] * v->e[j];
			}
		}
		return true;
	}


	/**
	 * c = v*w'
	 *
	 * @param c the product matrix.
	 * @param v the multiplicand vector.
	 * @param w the multiplier vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool mul(mat* c, const vec* v, const vec* w) {
		for(u32 i = 0; i < c->n_col; i++) {
			for(u32 j = 0; j < c->n_row; j++) {
				c->col[i]->e[j] = v->e[j] * w->e[i];
			}
		}
		return true;
	}


	/**
	 * d = sum(v.*v).
	 *
	 * @param d the product number.
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool inner(float* d, const vec* v) {
		*d = 0;
		for(u32 i = 0; i < v->n; i++) {
			*d += v->e[i] * v->e[i];
		}
		return true;
	}


	/**
	 * d = sum(v.*w).
	 *
	 * @param v the multiplicand vector.
	 * @param w the multiplier vector.
	 * @param d the product number.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool inner(float* d, const vec* v, const vec* w) {
		if(v->n != w->n) {
			printf("(inner: vector) not aligned!\n");
			return false;
		}
		*d = 0;
		for(u32 i = 0; i < v->n; i++) {
			*d += v->e[i] * w->e[i];
		}
		return true;
	}


	/**
	 * u = sum(a.*a).
	 *
	 * @param u the product vector.
	 * @param a the matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool inner(vec* u, const mat* a) {
		if(u->n != a->n_col) {
			printf("(inner: matrix) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			u->e[i] = 0;
			for(u32 j = 0; j < a->n_row; j++) {
				u->e[i] += a->col[i]->e[j] * a->col[i]->e[j];
			}
		}
		return true;
	}


	/**
	 * u' = sum(a.*b).
	 *
	 * @param u the product vector.
	 * @param a the multiplicand matrix.
	 * @param b the multiplier matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool inner(vec* u, const mat* a, const mat* b) {
		if(u->n != a->n_col || a->n_col != b->n_col || a->n_row != b->n_row) {
			printf("(inner: matrix) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < a->n_col; i++) {
			u->e[i] = 0;
			for(u32 j = 0; j < a->n_row; j++) {
				u->e[i] += a->col[i]->e[j] * b->col[i]->e[j];
			}
		}
		return true;
	}


	/**
	 * d = norm(v, 2).
	 *
	 * @param d the Euclidean norm.
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool norm(float* d, const vec* v) {
		inner(d, v);
		*d = sqrt(*d);
		return true;
	}


	/**
	 * Add a new entry at the end.
	 *
	 * @param v the vector.
	 * @param d the new entry.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool insert(vec* v, const float d) {
		v->n++;
		float* temp = (float*)malloc(v->n * sizeof(float));
		memcpy(temp, v->e, (v->n-1) * sizeof(float));
		free(v->e);
		v->e = temp;
		v->e[v->n-1] = d;
		return true;
	}


	/**
	 * Add a new entry at the end.
	 *
	 * @param x the index.
	 * @param i the new entry.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool insert(idx* x, const u32 i) {
		x->n++;
		u32* temp = (u32*)malloc(x->n * sizeof(u32));
		memcpy(temp, x->e, (x->n-1) * sizeof(u32));
		free(x->e);
		x->e = temp;
		x->e[x->n-1] = i;
		return true;
	}


	/**
	 * Add a new row at the end.
	 *
	 * @param a the matrix.
	 * @param v the new vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool insert_row(mat* a, const vec* v) {
		if(a->n_col != v->n) {
			printf("(insert_row) not aligned!\n");
			return false;
		}
		a->n_row++;
		float* temp;
		for(u32 i = 0; i < a->n_col; i++) {
			a->col[i]->n++;
			temp = (float*)malloc(a->n_row * sizeof(float));
			memcpy(temp, a->col[i]->e, (a->n_row-1) * sizeof(float));
			free(a->col[i]->e);
			a->col[i]->e = temp;
			a->col[i]->e[a->n_row-1] = v->e[i];
		}
		return true;
	}


	/**
	 * Add a new column at the end.
	 *
	 * @param a the matrix.
	 * @param v the new vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool insert_col(mat* a, const vec* v) {
		if(a->n_row != v->n) {
			printf("(insert_col) not aligned!\n");
			return false;
		}
		a->n_col++;
		vec** temp = (vec**)malloc(a->n_col * sizeof(vec*));
		memcpy(temp, a->col, (a->n_col-1) * sizeof(vec*));
		free(a->col);
		a->col = temp;
		a->col[a->n_col-1] = new vec(v->n);
		copy(a->col[a->n_col-1], v);
		return true;
	}


	/**
	 * Add a new row and a new column at the end.
	 *
	 * @param a the matrix.
	 * @param d the number.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool insert(mat* a, const float d) {
		a->n_row++;
		a->n_col++;
		float* ftemp;
		for(u32 i = 0; i < a->n_col-1; i++) {
			a->col[i]->n++;
			ftemp = (float*)malloc(a->n_row * sizeof(float));
			memcpy(ftemp, a->col[i]->e, (a->n_row-1) * sizeof(float));
			free(a->col[i]->e);
			a->col[i]->e = ftemp;
			a->col[i]->e[a->n_row-1] = d;
		}
		vec** temp = (vec**)malloc(a->n_col * sizeof(vec*));
		memcpy(temp, a->col, (a->n_col-1) * sizeof(vec*));
		free(a->col);
		a->col = temp;
		a->col[a->n_col-1] = new vec(a->n_row, d);
		return true;
	}


	/**
	 * Remove the last entry.
	 *
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool shed(vec* v) {
		if(v->n == 0) {
			printf("(shed: vector) empty!\n");
			return false;
		}
		v->n--;
		if(v->n == 0) {
			free(v->e);
			v->e = 0;
			return true;
		}
		float* temp = (float*)malloc(v->n * sizeof(float));
		memcpy(temp, v->e, v->n * sizeof(float));
		free(v->e);
		v->e = temp;
		return true;
	}


	/**
	 * Remove the last entry.
	 *
	 * @param x the index.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool shed(idx* x) {
		if(x->n == 0) {
			printf("(shed: index) empty!\n");
			return false;
		}
		x->n--;
		if(x->n == 0) {
			free(x->e);
			x->e = 0;
			return true;
		}
		u32* temp = (u32*)malloc(x->n * sizeof(u32));
		memcpy(temp, x->e, x->n * sizeof(u32));
		free(x->e);
		x->e = temp;
		return true;
	}


	/**
	 * Remove the last n entries.
	 *
	 * @param x the index.
	 * @param n number of entries.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool shed(idx* x, const u32 n) {
		if(n == 0) {
			return true;
		}
		if(x->n < n) {
			printf("(shed: index) n too large!\n");
			return false;
		}
		x->n -= n;
		if(x->n == 0) {
			free(x->e);
			x->e = 0;
			return true;
		}
		u32* temp = (u32*)malloc(x->n * sizeof(u32));
		memcpy(temp, x->e, x->n * sizeof(u32));
		free(x->e);
		x->e = temp;
		return true;
	}


	/**
	 * Remove the last row.
	 *
	 * @param a the matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool shed_row(mat* a) {
		if(a->n_row == 0) {
			printf("(shed_row) empty!\n");
			return false;	
		}
		a->n_row--;
		if(a->n_row == 0) {
			for(u32 i = 0; i < a->n_col; i++) {
				a->col[i]->n = 0;
				free(a->col[i]->e);
				a->col[i]->e = 0;
			}
			return true;
		}
		float* temp;
		for(u32 i = 0; i < a->n_col; i++) {
			a->col[i]->n--;
			temp = (float*)malloc(a->n_row * sizeof(float));
			memcpy(temp, a->col[i]->e, a->n_row * sizeof(float));
			free(a->col[i]->e);
			a->col[i]->e = temp;
		}
		return true;
	}


	/**
	 * Remove the last column.
	 *
	 * @param a the matrix.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool shed_col(mat* a) {
		if(a->n_col == 0) {
			printf("(shed_col) empty!\n");
			return false;
		}
		a->n_col--;
		free(a->col[a->n_col]);
		if(a->n_col == 0) {
			free(a->col);
			a->col = 0;
			return true;
		}
		vec** temp = (vec**)malloc(a->n_col * sizeof(vec*));
		memcpy(temp, a->col, a->n_col * sizeof(vec*));
		free(a->col);
		a->col = temp;
		return true;
	}


	/**
	 * Swap two entries.
	 *
	 * @param v the vector.
	 * @param i the index of first entry.
	 * @param j the index of second entry.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool swap(vec* v, const u32 i, const u32 j) {
		float temp;
		temp = v->e[i];
		v->e[i] = v->e[j];
		v->e[j] = temp;
		return true;
	}


	/**
	 * Swap two entries.
	 *
	 * @param x the index.
	 * @param i the index of first entry.
	 * @param j the index of second entry.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool swap(idx* x, const u32 i, const u32 j) {
		u32 temp;
		temp = x->e[i];
		x->e[i] = x->e[j];
		x->e[j] = temp;
		return true;
	}


	/**
	 * Swap two rows.
	 *
	 * @param a the matrix.
	 * @param i the index of first row.
	 * @param j the index of second row.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool swap_row(mat* a, const u32 i, const u32 j) {
		float temp;
		for(u32 k = 0; k < a->n_col; k++) {
			temp = a->col[k]->e[i];
			a->col[k]->e[i] = a->col[k]->e[j];
			a->col[k]->e[j] = temp;
		}
		return true;
	}


	/**
	 * Swap two columns.
	 *
	 * @param a the matrix.
	 * @param i the index of first column.
	 * @param j the index of second column.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool swap_col(mat* a, const u32 i, const u32 j) {
		vec* temp;
		temp = a->col[i];
		a->col[i] = a->col[j];
		a->col[j] = temp;
		return true;
	}


	/**
	 * Find the index of target element
	 *
	 * @param k the index.
	 * @param x the vector.
	 * @param i the element.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool find_index(u32* k, const idx* x, const u32 i) {
		for(u32 j = 0; j < x->n; j++) {
			if(x->e[j] == i) {
				*k = j;
				return true;
			}
		}
		*k = UINT32_MAX;
		printf("(find_index) index not found!\n");
		return false;
	}


	/**
	 * Find the index of minimal element
	 *
	 * @param k the index.
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool find_min_index(u32* k, const vec* v) {
		float d = _FPCLASS_PINF;
		for(u32 j = 0; j < v->n; j++) {
			if(v->e[j] < d) {
				*k = j;
			}
		}
		return true;
	}


	/**
	 * Sort a index in ascending order
	 *
	 * @param x the index.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool sort_ascend(idx* x) {
		u32 temp;
		for(u32 i = x->n-1; i > 0; i--) {
			for(u32 j = 0; j < i; j++) {
				if(x->e[j] > x->e[j+1]) {
					temp = x->e[j];
					x->e[j] = x->e[j+1];
					x->e[j+1] = temp;
				}
			}
		}
		return true;
	}


	/**
	 * Sort a index in descending order
	 *
	 * @param x the index.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool sort_descend(idx* x) {
		u32 temp;
		for(u32 i = x->n-1; i > 0; i--) {
			for(u32 j = 0; j < i; j++) {
				if(x->e[j] < x->e[j+1]) {
					temp = x->e[j];
					x->e[j] = x->e[j+1];
					x->e[j+1] = temp;
				}
			}
		}
		return true;
	}


	/**
	 * Sort index of a vector in ascending order
	 *
	 * @param z the sorted index.
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool sort_index_ascend(idx* z, const vec* v) {
		if(z->n != v->n) {
			printf("(sort_index) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < v->n; i++) {
			z->e[i] = i;
		}
		if(v->n < 2) {
			return true;
		}
		vec* u = new vec(v->n);
		copy(u, v);
		float utemp;
		u32 xtemp;
		for(u32 i = u->n-1; i > 0; i--) {
			for(u32 j = 0; j < i; j++) {
				if(u->e[j] > u->e[j+1]) {
					utemp = u->e[j];
					u->e[j] =u->e[j+1];
					u->e[j+1] = utemp;
					xtemp = z->e[j];
					z->e[j] = z->e[j+1];
					z->e[j+1] = xtemp;
				}
			}
		}
		delete u;
		return true;
	}


	/**
	 * Sort index of a vector in descending order
	 *
	 * @param z the sorted index.
	 * @param v the vector.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool sort_index_descend(idx* z, const vec* v) {
		if(z->n != v->n) {
			printf("(sort_index_descend) not aligned!\n");
			return false;
		}
		for(u32 i = 0; i < v->n; i++) {
			z->e[i] = i;
		}
		if(v->n < 2) {
			return true;
		}
		vec* u = new vec(v->n);
		copy(u, v);
		float utemp;
		u32 xtemp;
		for(u32 i = u->n-1; i > 0; i--) {
			for(u32 j = 0; j < i; j++) {
				if(u->e[j] < u->e[j+1]) {
					utemp = u->e[j];
					u->e[j] =u->e[j+1];
					u->e[j+1] = utemp;
					xtemp = z->e[j];
					z->e[j] = z->e[j+1];
					z->e[j+1] = xtemp;
				}
			}
		}
		delete u;
		return true;
	}


	/**
	 * Set complement of a sorted index
	 *
	 * @param z the complement set index.
	 * @param x the origin set index.
	 * @param n the length of universe.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool complement(idx* z, const idx* x, const u32 n) {
		if(z->n + x->n != n) {
			printf("(complement) not aligned!\n");
			return false;
		}
		u32 i, j, d;
		for(i = 0, j = 0, d = 0; j < z->n; d++) {
			if(i < x->n && d == x->e[i]) {
				i++;
			}
			else {
				z->e[j] = d;
				j++;
			}
		}
		return true;
	}


	/**
	 * Set difference of two sorted index
	 *
	 * @param z the difference set index.
	 * @param x the minuend set index.
	 * @param y the subtrahend set index.
	 * @return whether this function has been executed successfully.
	 */
	__host__ __device__ bool set_difference(idx* z, const idx* x, const idx* y) {
		if(z->n != x->n) {
			printf("(set_difference) not aligned!\n");
			return false;
		}
		u32 i, j, k;
		for(i = 0, j = 0, k = 0; i < x->n && j < y->n;) {
			if(x->e[i] < y->e[j]) {
				z->e[k] = x->e[i] ;
				i++;
				k++;
			}else if(x->e[i] == y->e[j]) {
				i++;
				j++;
			}else {
				j++;
			}
		}
		while(k != z->n) {
			shed(z, z->n - k);
		}
		return true;
	}
}
